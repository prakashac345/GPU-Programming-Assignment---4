#include "hip/hip_runtime.h"
//%%writefile main.cu
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define max_N 100000
#define max_P 30
#define BLOCKSIZE 1024

using namespace std;

struct Request
{
    int RequestId;
    int facility;
    int centre;
    int start;
    int slots;
};

//*******************************************

// Write down the kernels here
__global__ void CheckSlot(int index, int startSlot, int endSlot, int *slots, int &flag)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid + startSlot >= endSlot)
        return;
    if (slots[index * 24 + tid + startSlot - 1] - 1 < 0)
    {
        atomicAdd(&flag, 1);
    }
}

__global__ void UpdateSlot(int index, int startSlot, int endSlot, int *slots)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid + startSlot >= endSlot)
        return;
    slots[index * 24 + tid + startSlot - 1]--;
}

__global__ void setSlots(int NF, int *slots, int *capacity)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= NF)
        return;
    tid = tid * 24;
    for (int i = 0; i < 24; i++)
    {
        slots[tid + i] = capacity[tid / 24];
    }
}

__global__ void CountFacilityKernel(int tidCenter, int *slots, int *flags, int *PrefixNoReqPerFac, int *NoReqPerFac, struct Request *sortR, int *succ_reqs)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int index = tidCenter * max_P + tid;

    if (NoReqPerFac[index] == 0)
        return;

    int NoReq;
    if (index >= 1)
        NoReq = PrefixNoReqPerFac[index - 1];
    else
        NoReq = 0;

    int NoReqNext = PrefixNoReqPerFac[index];

    for (int i = NoReq; i < NoReqNext; i++)
    {
        struct Request R = sortR[i];
        int req_cenx = R.centre;
        int req_facx = R.facility;
        int req_startx = R.start;
        int req_slotsx = R.slots;

        if (req_facx == tid && req_cenx == tidCenter)
        {
            flags[index] = 0;

            CheckSlot<<<(req_slotsx + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE>>>(index, req_startx, req_startx + req_slotsx, slots, flags[index]);
            hipDeviceSynchronize();

            if (flags[index] == 0)
            {
                UpdateSlot<<<(req_slotsx + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE>>>(index, req_startx, req_startx + req_slotsx, slots);
                hipDeviceSynchronize();

                atomicAdd(&succ_reqs[tidCenter], 1);
            }
        }
    }
}

__global__ void CountSuccessKernel(int N, int *slots, int *flags, int *PrefixNoReqPerFac, int *NoReqPerFac, int *succ_reqs, int *facility, struct Request *sortR)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= N)
        return;

    int NoFac = facility[tid];
    CountFacilityKernel<<<(NoFac + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE>>>(tid, slots, flags, PrefixNoReqPerFac, NoReqPerFac, sortR, succ_reqs);
}
__global__ void NoReqPerFacKernel(int R, int *d_NoReqPerFac, struct Request *sortR)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= R)
        return;

    struct Request R1 = sortR[tid];
    int req_cenx = R1.centre;
    int req_facx = R1.facility;
    atomicAdd(&d_NoReqPerFac[req_cenx * max_P + req_facx], 1);
}

//***********************************************

// compare function for qsort to sort requests by centre and facility and then by id
int comp(const void *a, const void *b)
{
    Request *r1 = (Request *)a;
    Request *r2 = (Request *)b;
    if (r1->centre < r2->centre)
        return -1;
    if (r1->centre > r2->centre)
        return 1;
    if (r1->facility < r2->facility)
        return -1;
    if (r1->facility > r2->facility)
        return 1;
    if (r1->RequestId < r2->RequestId)
        return -1;
    if (r1->RequestId > r2->RequestId)
        return 1;
    return 0;
}

int main(int argc, char **argv)
{
    // variable declarations...
    int N, *centre, *facility, *capacity, *fac_ids, *succ_reqs, *tot_reqs;

    FILE *inputfilepointer;

    // File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer = fopen(inputfilename, "r");

    if (inputfilepointer == NULL)
    {
        printf("input.txt file failed to open.");
        return 0;
    }

    fscanf(inputfilepointer, "%d", &N); // N is number of centres

    // Allocate memory on cpu
    centre = (int *)malloc(N * sizeof(int));           // Computer  centre numbers
    facility = (int *)malloc(N * sizeof(int));         // Number of facilities in each computer centre
    fac_ids = (int *)malloc(max_P * N * sizeof(int));  // Facility room numbers of each computer centre
    capacity = (int *)malloc(max_P * N * sizeof(int)); // stores capacities of each facility for every computer centre

    int success = 0;                            // total successful requests
    int fail = 0;                               // total failed requests
    tot_reqs = (int *)malloc(N * sizeof(int));  // total requests for each centre
    succ_reqs = (int *)malloc(N * sizeof(int)); // total successful requests for each centre

    // Input the computer centres data
    int k1 = 0, k2 = 0;
    for (int i = 0; i < N; i++)
    {
        fscanf(inputfilepointer, "%d", &centre[i]);
        fscanf(inputfilepointer, "%d", &facility[i]);

        for (int j = 0; j < facility[i]; j++)
        {
            fscanf(inputfilepointer, "%d", &fac_ids[k1]);
            k1++;
        }
        for (int j = 0; j < facility[i]; j++)
        {
            fscanf(inputfilepointer, "%d", &capacity[k2]);
            k2++;
        }
        for (int j = 0; j < max_P - facility[i]; j++)
        {
            fac_ids[k1] = -1;
            capacity[k2] = 0;
            k1++;
            k2++;
        }
    }

    // variable declarations
    int *req_id, *req_cen, *req_fac, *req_start, *req_slots; // Number of slots requested for every request

    // Allocate memory on CPU
    int R;
    fscanf(inputfilepointer, "%d", &R);           // Total requests
    req_id = (int *)malloc((R) * sizeof(int));    // Request ids
    req_cen = (int *)malloc((R) * sizeof(int));   // Requested computer centre
    req_fac = (int *)malloc((R) * sizeof(int));   // Requested facility
    req_start = (int *)malloc((R) * sizeof(int)); // Start slot of every request
    req_slots = (int *)malloc((R) * sizeof(int)); // Number of slots requested for every request

    // Input the user request data
    for (int j = 0; j < R; j++)
    {
        fscanf(inputfilepointer, "%d", &req_id[j]);
        fscanf(inputfilepointer, "%d", &req_cen[j]);
        fscanf(inputfilepointer, "%d", &req_fac[j]);
        fscanf(inputfilepointer, "%d", &req_start[j]);
        fscanf(inputfilepointer, "%d", &req_slots[j]);
        tot_reqs[req_cen[j]] += 1;
    }

    // Sorting the requests by centre and facility and then by id
    struct Request *reqSortR = (struct Request *)malloc((R) * sizeof(struct Request));
    for (int i = 0; i < R; i++)
    {
        reqSortR[i].RequestId = req_id[i];
        reqSortR[i].facility = req_fac[i];
        reqSortR[i].centre = req_cen[i];
        reqSortR[i].start = req_start[i];
        reqSortR[i].slots = req_slots[i];
    }

    qsort(reqSortR, R, sizeof(struct Request), comp);

    int *NoReqPerFac, *d_NoReqPerFac;
    struct Request *d_reqSortR;
    NoReqPerFac = (int *)malloc((max_P * N) * sizeof(int));
    hipMalloc(&d_NoReqPerFac, (max_P * N) * sizeof(int));
    hipMalloc(&d_reqSortR, R * sizeof(struct Request));

    hipMemset(d_NoReqPerFac, 0, (max_P * N) * sizeof(int));
    hipMemcpy(d_reqSortR, reqSortR, R * sizeof(struct Request), hipMemcpyHostToDevice);

    // allocate memory on GPU
    int *d_facility, *d_capacity;
    hipMalloc(&d_facility, N * sizeof(int));
    hipMalloc(&d_capacity, max_P * N * sizeof(int));
    hipMemcpy(d_facility, facility, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_capacity, capacity, max_P * N * sizeof(int), hipMemcpyHostToDevice);

    int *succ_reqs_d;
    hipMalloc(&succ_reqs_d, N * sizeof(int));

    int *d_slots;
    hipMalloc(&d_slots, 24 * max_P * N * sizeof(int));

    int *d_flags;
    hipMalloc(&d_flags, max_P * N * sizeof(int));
    hipMemset(d_flags, 0, max_P * N * sizeof(int));

    //*********************************
    // Call the kernels here

    setSlots<<<(N * max_P - 1) / BLOCKSIZE + 1, BLOCKSIZE>>>(N * max_P, d_slots, d_capacity);
    hipDeviceSynchronize();

    dim3 dimGrid((R - 1) / BLOCKSIZE + 1, 1, 1);
    dim3 dimBlock(BLOCKSIZE, 1, 1);
    NoReqPerFacKernel<<<dimGrid, dimBlock>>>(R, d_NoReqPerFac, d_reqSortR);
    hipMemcpy(NoReqPerFac, d_NoReqPerFac, max_P * N * sizeof(int), hipMemcpyDeviceToHost);

    int *PrefixNoReqPerFac;
    PrefixNoReqPerFac = (int *)malloc((max_P * N) * sizeof(int));
    PrefixNoReqPerFac[0] = NoReqPerFac[0];
    for (int i = 1; i < max_P * N; i++)
    {
        PrefixNoReqPerFac[i] = PrefixNoReqPerFac[i - 1] + NoReqPerFac[i];
    }

    int *d_PrefixNoReqPerFac;
    hipMalloc(&d_PrefixNoReqPerFac, max_P * N * sizeof(int));
    hipMemcpy(d_PrefixNoReqPerFac, PrefixNoReqPerFac, max_P * N * sizeof(int), hipMemcpyHostToDevice);

    CountSuccessKernel<<<(N - 1) / BLOCKSIZE + 1, BLOCKSIZE>>>(N, d_slots, d_flags, d_PrefixNoReqPerFac, d_NoReqPerFac, succ_reqs_d, d_facility, d_reqSortR);
    hipMemcpy(succ_reqs, succ_reqs_d, N * sizeof(int), hipMemcpyDeviceToHost);

    int total = R;
    for (int i = 0; i < N; i++)
    {
        success += succ_reqs[i];
    }

    fail = total - success;
    //********************************

    // Output
    char *outputfilename = argv[2];
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename, "w");

    fprintf(outputfilepointer, "%d %d\n", success, fail);
    for (int j = 0; j < N; j++)
    {
        fprintf(outputfilepointer, "%d %d\n", succ_reqs[j], tot_reqs[j] - succ_reqs[j]);
    }
    fclose(inputfilepointer);
    fclose(outputfilepointer);
    hipDeviceSynchronize();
    return 0;
}